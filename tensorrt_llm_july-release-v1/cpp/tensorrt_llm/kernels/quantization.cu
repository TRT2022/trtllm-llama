#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/int8Utils.cuh"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/quantization.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

__global__ void quantizedKernel(char4* dst, const float4* src, const int64_t sizeDiv4, const float* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        const float scale = __ldg(scalePtr);
        char4 tmp;
        const float4 floatTmp = __ldg(src + idx);
        tmp.x = cuda_cast<int8_t>(floatTmp.x * scale);
        tmp.y = cuda_cast<int8_t>(floatTmp.y * scale);
        tmp.z = cuda_cast<int8_t>(floatTmp.z * scale);
        tmp.w = cuda_cast<int8_t>(floatTmp.w * scale);
        dst[idx] = tmp;
    }
}

__global__ void quantizedKernel(char4* dst, const half2* src, const int64_t sizeDiv4, const float* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        const float scale = __ldg(scalePtr);
        char4 tmp;
        int srcId = idx << 1;

        const uint2 h2 = __ldg(reinterpret_cast<const uint2*>(src + srcId));

        const half2 half2Tmp = reinterpret_cast<const half2&>(h2.x);
        const half2 half2Tmp2 = reinterpret_cast<const half2&>(h2.y);

        tmp.x = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.x) * scale);
        tmp.y = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.y) * scale);
        tmp.z = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.x) * scale);
        tmp.w = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.y) * scale);
        dst[idx] = tmp;
    }
}

template <typename T>
void invokeQuantization(
    int8_t* dst, const T* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize)
{
    TLLM_CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

    int numBlocks{static_cast<int>((size + 255) / 256)};
    dim3 grid(std::min(numBlocks, maxGridSize));
    TLLM_CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
    dim3 block(64);
    if (std::is_same_v<T, float>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (const float4*) src, size / 4, scalePtr);
    }
    else if (std::is_same_v<T, half>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (const half2*) src, size / 4, scalePtr);
    }
}

template void invokeQuantization<float>(
    int8_t* dst, const float* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize);

template void invokeQuantization<half>(
    int8_t* dst, const half* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize);

template <typename T>
__global__ void perTokenQuantization(
    int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr)
{
    const T* srcRow = src + blockIdx.x * numCols;
    int8_t* dstRow = dst + blockIdx.x * numCols;

    T localMax = 1e-6f;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x)
    {
        localMax = cuda_max(localMax, cuda_abs(srcRow[i]));
    }
    const float rowMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0)
    {
        scalePtr[blockIdx.x] = rowMax / 127.f;
    }

    const float scaleOrigQuant = 127.f / rowMax;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x)
    {
        dstRow[i] = cuda_cast<int8_t>(cuda_cast<float>(srcRow[i]) * scaleOrigQuant);
    }
}

template <typename T>
void invokePerTokenQuantization(
    int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr, hipStream_t stream)
{
    // each block is responsible for a single row
    const dim3 block(512);
    const dim3 grid(numRows);

    perTokenQuantization<<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr);
}

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T)                                                                   \
    template void invokePerTokenQuantization(                                                                          \
        int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr, hipStream_t stream)

INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16);
#endif

} // namespace kernels
} // namespace tensorrt_llm
