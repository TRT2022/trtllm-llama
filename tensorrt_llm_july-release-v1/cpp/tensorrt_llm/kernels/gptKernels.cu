#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "tensorrt_llm/common/cudaBf16Wrapper.h"
#include "tensorrt_llm/common/cudaFp8Utils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/gptKernels.h"
#include <hipcub/hipcub.hpp>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

// A stateful callback functor that maintains the running sum between consecutive scans.
struct BlockPrefixCallbackOp
{
    // Running prefix
    int mRunningTotal;

    // Constructor
    __device__ BlockPrefixCallbackOp(int runningTotal)
        : mRunningTotal(runningTotal)
    {
    }

    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ int operator()(int blockAggregate)
    {
        int oldPrefix = mRunningTotal;
        mRunningTotal += blockAggregate;
        return oldPrefix;
    }
};

// Given an array of sequence lengths, with batchSize elements, that kernel computes the exclusive
// prefix-sums of the sequence lengths. There are (batchSize+1) elements in seqOffsets.
//
// seqOffsets[ 0]        = 0
// seqOffsets[ii]        = seqLengths[0] + .. + seqLengths[ii-1],
// seqOffsets[batchSize] = seqLengths[0] + .. + seqLengths[batchSize-1]
//
// This kernel uses a single thread block of THREADS_PER_BLOCK threads.

template <int THREADS_PER_BLOCK>
__global__ __launch_bounds__(THREADS_PER_BLOCK) void computeSeqOffsets(
    int* seqOffsets, const int* seqLengths, int batchSize)
{
    // The implementation of the parallel scan in the thread block (see CUB for details).
    using BlockScan = hipcub::BlockScan<int, THREADS_PER_BLOCK>;

    // Allocate storage in shared memory to do the scan.
    __shared__ typename BlockScan::TempStorage tempStorage;

    // This prefixOp operator keeps a running sum for when we need multiple iterations of the loop.
    BlockPrefixCallbackOp prefixOp(0);

    // Iterate over the sequences in the batch.
    //
    // The loop index does not depend on the thread index to make sure all the threads enter the
    // loop as we have __syncthreads in it (and we need all threads to participate to avoid
    // deadlocks).
    for (int batchOffset = 0; batchOffset <= batchSize; batchOffset += THREADS_PER_BLOCK)
    {
        // The index of the batch.
        int batchIdx = batchOffset + threadIdx.x;

        // Threads that correspond to valid sequences read the length.
        int seqLength = 0;
        if (batchIdx < batchSize)
        {
            seqLength = seqLengths[batchIdx];
        }

        // Do the prefix-scan (it calls syncthreads internally).
        int seqOffset;
        BlockScan(tempStorage).ExclusiveSum(seqLength, seqOffset, prefixOp);

        // Store the result.
        if (batchIdx <= batchSize)
        {
            seqOffsets[batchIdx] = seqOffset;
        }

        // Make sure the shared memory can be reused for the next iteration of the loop.
        __syncthreads();
    }
}

// This kernel computes the padding offsets: Given the index (idx) of a token in a ragged tensor,
// we need the index of the token in the corresponding tensor with padding. We compute an array
// of numTokens elements, called the paddingOffsets, such that the position in the padded tensor
// of the token "idx" in the ragged tensor is given by idx + paddingOffset[idx].
//
// That kernel uses a grid of batchSize blocks.

__global__ void computePaddingOffsets(int* paddingOffsets, const int* seqOffsets, int maxSeqLength)
{
    // The index of the sequence in the batch.
    int batchIdx = blockIdx.x;

    // The beginning of the sequence.
    int seqBegin = seqOffsets[batchIdx];
    // The offset to the 1st element of the next sequence.
    int seqEnd = seqOffsets[batchIdx + 1];
    // The length of the sequence.
    int seqLength = seqEnd - seqBegin;

    // The number of padded tokens in the previous sequences.
    int paddingOffset = batchIdx * maxSeqLength - seqBegin;

    // Iterate over the tokens to update the number of padded elements.
    for (int tokenIdx = threadIdx.x; tokenIdx < seqLength; tokenIdx += blockDim.x)
    {
        paddingOffsets[seqBegin + tokenIdx] = paddingOffset + max(0, tokenIdx - seqLength);
    }
}

// This kernel computes the attention mask. We must compute this on-the-fly in the future.

template <typename AttentionMaskDataType>
__global__ void computeAttentionMask(
    AttentionMaskDataType* attentionMask, const int* seqOffsets, int maxSeqLength, AttentionMaskType attentionMaskType)
{
    // The index of the sequence in the batch.
    int batchIdx = blockIdx.y;

    // The number of items in the mask for each sequence.
    int maskSize = maxSeqLength * maxSeqLength;
    // The offset to the 1st element of the mask for that particular sequence.
    int batchOffset = batchIdx * maskSize;

    // The beginning of the sequence.
    int seqBegin = seqOffsets[batchIdx];
    // The offset to the 1st element of the next sequence.
    int seqEnd = seqOffsets[batchIdx + 1];
    // The length of the sequence.
    int seqLength = seqEnd - seqBegin;

    // Iterate over the tokens to update the number of padded elements.
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < maskSize; idx += blockDim.x)
    {
        // The position in the matrix.
        int rowIdx = idx / maxSeqLength;
        int colIdx = idx % maxSeqLength;

        // Is it a valid token?
        bool isValid = true;
        switch (attentionMaskType)
        {
        case AttentionMaskType::PADDING:
            isValid = rowIdx < seqLength && colIdx < seqLength;
            // seq_length==4, max_seq_len==5
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 0 0 0 0 0
            break;
        case AttentionMaskType::CAUSAL:
            isValid = rowIdx < seqLength && colIdx < seqLength && colIdx <= rowIdx;
            // seq_length==4, max_seq_len==5
            // 1 0 0 0 0
            // 1 1 0 0 0
            // 1 1 1 0 0
            // 1 1 1 1 0
            // 0 0 0 0 0
            break;
        case AttentionMaskType::BIDIRECTIONAL:
            // clang-format off
            isValid = (rowIdx <  seqLength - 1 && colIdx < seqLength - 1) ||
                      (rowIdx == seqLength - 1 && colIdx < seqLength);
            // clang-format on
            // seq_length==4, max_seq_len==5, only use in context phase
            // 1 1 1 0 0
            // 1 1 1 0 0
            // 1 1 1 0 0
            // 1 1 1 1 0
            // 0 0 0 0 0
            break;
        }

        // Store the mask.
        attentionMask[batchOffset + idx] = isValid ? AttentionMaskDataType(1) : AttentionMaskDataType(0);
    }
}

template <typename T>
void invokeBuildDecoderInfo(const BuildDecoderInfoParams<T>& params, hipStream_t stream)
{
    // Compute the sequence offsets.
    const int THREADS_PER_BLOCK = 256;
    computeSeqOffsets<THREADS_PER_BLOCK>
        <<<1, THREADS_PER_BLOCK, 0, stream>>>(params.seqOffsets, params.seqLengths, params.batchSize);

    // Compute the padding offsets.
    computePaddingOffsets<<<params.batchSize, THREADS_PER_BLOCK, 0, stream>>>(
        params.paddingOffsets, params.seqOffsets, params.maxSeqLength);

    // Compute the attention mask, if needed.
    if (params.attentionMask != nullptr)
    {
        // large value like 512 hurts kernel perf at long sequence length. Keep small for now.
        const int MIN_BLOCKS = 16;
        int blocksPerSeq = 16;
        while (blocksPerSeq * params.batchSize < MIN_BLOCKS)
        {
            blocksPerSeq *= 2;
        }
        dim3 grid(blocksPerSeq, params.batchSize);
        computeAttentionMask<<<grid, THREADS_PER_BLOCK, 0, stream>>>(
            params.attentionMask, params.seqOffsets, params.maxSeqLength, params.attentionMaskType);
    }
}

template void invokeBuildDecoderInfo(const BuildDecoderInfoParams<float>&, hipStream_t);
template void invokeBuildDecoderInfo(const BuildDecoderInfoParams<half>&, hipStream_t);
#ifdef ENABLE_BF16
template void invokeBuildDecoderInfo(const BuildDecoderInfoParams<__hip_bfloat16>&, hipStream_t);
#endif
#ifdef ENABLE_FP8
template void invokeBuildDecoderInfo(const BuildDecoderInfoParams<__hip_fp8_e4m3_fnuz>&, hipStream_t);
#endif

__global__ void updatePaddingCountKernel(int* paddingPerSeq, const int* seqLengths, int maxSeqLength, int batchSize)
{

    for (int ii = threadIdx.x; ii < batchSize; ii += blockDim.x)
    {
        paddingPerSeq[ii] = maxSeqLength - seqLengths[ii];
    }
}

void invokeUpdatePaddingCount(
    int* paddingPerSeq, const int* seqLengths, int maxSeqLength, int batchSize, hipStream_t stream)
{
    int blockSize = min(1024, batchSize);
    updatePaddingCountKernel<<<1, blockSize, 0, stream>>>(paddingPerSeq, seqLengths, maxSeqLength, batchSize);
}

} // namespace kernels
} // namespace tensorrt_llm
