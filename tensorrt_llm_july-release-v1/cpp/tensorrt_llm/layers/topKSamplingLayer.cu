#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/topKSamplingLayer.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

template <uint32_t TOP_K_MAX>
__global__ void setup_topk_runtime_args(int batch_size, uint32_t top_k, uint32_t* top_ks, int top_ks_size, float top_p,
    float* top_ps, int top_ps_size, bool* skip_decode)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < batch_size; i += gridDim.x * blockDim.x)
    {
        uint32_t k = top_ks_size > 1 ? top_ks[i] : top_k;
        float p = top_ps_size > 1 ? top_ps[i] : top_p;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        if (k > 0 && p == 0.0f)
        {
            // for compatibility <= TensorRT-LLM5.0.
            // This case corresponds to the old topk sampling, which is equivalent to
            // the old topk_topp sampling with topp=1.0f. TopKSamplingLayer and
            // TopKTopPSamplingLayer are now merged by TopKSamplingLayer. Thus, we
            // replace the case topk>0 and topp=0.0f by topk>0 and topp=1.0f for the
            // compatibility.
            p = 1.0f;
        }
        // Clip k value. A topk sampling kernel supports up to TOP_K_MAX=64.
        top_ks[i] = k > TOP_K_MAX ? TOP_K_MAX : k;
        if (k > TOP_K_MAX)
        {
            printf(
                "[WARNING] topk (%d) is larger than max supported number (%d) for "
                "token %d"
                " clip to max supported number %d. \n",
                k, TOP_K_MAX, i, top_ks[i]);
        }
        // Clip p value if it is out of range. range = [0.0, 1.0].
        top_ps[i] = p < 0.0f ? 0.0f : (p > 1.0f ? 1.0f : p);
        if (p < 0.0f || p > 1.0f)
        {
            printf(
                "[WARNING] topp (%f) is out of range ([0.0, 1.0f]) for token %d"
                " clip to closest number %f.\n",
                p, i, top_ps[i]);
        }
        skip_decode[i] = k == 0;
    }
}

template <typename T>
void TopKSamplingLayer<T>::allocateBuffer(size_t const batch_size, std::vector<uint32_t> const& top_k)
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
    uint32_t max_top_k = (top_k.size() > 0) ? *std::max_element(std::begin(top_k), std::end(top_k)) : 1;
    if (max_top_k == 0)
    {
        // for safety. TopKSamplingLayer handles a case of top_k=0 and top_p=0 as
        // a greedy decode, i.e. top_k=1, although such case has max_top_k=0.
        max_top_k = 1;
    }
    invokeTopKSampling<T>(nullptr, sampling_workspace_size_, nullptr, nullptr, nullptr, nullptr, nullptr, nullptr,
        nullptr, max_top_k, 1.0f, vocab_size_padded_, nullptr, stream_, batch_size, skip_decode_buf_);
    sampling_workspace_ = allocator_->reMalloc(sampling_workspace_, sampling_workspace_size_, false);
    runtime_top_k_buf_ = allocator_->reMalloc(runtime_top_k_buf_, sizeof(uint32_t) * batch_size, false);
    runtime_top_p_buf_ = allocator_->reMalloc(runtime_top_p_buf_, sizeof(float) * batch_size, false);
    is_allocate_buffer_ = true;
}

template <typename T>
void TopKSamplingLayer<T>::freeBuffer()
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
    if (is_allocate_buffer_)
    {
        allocator_->free((void**) (&sampling_workspace_));
        allocator_->free((void**) (&runtime_top_k_buf_));
        allocator_->free((void**) (&runtime_top_p_buf_));
    }
    BaseSamplingLayer<T>::freeBuffer();
    is_allocate_buffer_ = false;
}

template <typename T>
void TopKSamplingLayer<T>::setup(size_t const batch_size, SetupParams const& setupParams)
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
    BaseSamplingLayer<T>::setupBase(batch_size, setupParams);

    uint32_t const default_top_k = 0;
    auto const runtime_top_k = setupParams.runtime_top_k.value_or(std::vector<uint32_t>{default_top_k});
    auto const runtime_top_p = setupParams.runtime_top_p.value_or(std::vector<float>{});

    allocateBuffer(batch_size, runtime_top_k);

    size_t const runtime_top_k_size = runtime_top_k.size();
    size_t const runtime_top_p_size = runtime_top_p.size();

    uint32_t const top_k = *std::max_element(std::begin(runtime_top_k), std::end(runtime_top_k));
    float const top_p = (runtime_top_p_size == 0) ? 0.0f : runtime_top_p.front();

    if (runtime_top_k_size > 1)
    {
        TLLM_CHECK_WITH_INFO(runtime_top_k.size() == batch_size,
            fmtstr(
                "runtime_top_k.size() (%lu) == batch_size (%lu) is not satisfied!", runtime_top_k.size(), batch_size));
        cudaAutoCpy(runtime_top_k_buf_, runtime_top_k.data(), batch_size, stream_);
    }
    if (runtime_top_p_size > 1)
    {
        TLLM_CHECK_WITH_INFO(runtime_top_p.size() == batch_size,
            fmtstr(
                "runtime_top_p.size() (%lu) == batch_size (%lu) is not satisfied!", runtime_top_p.size(), batch_size));
        cudaAutoCpy(runtime_top_p_buf_, runtime_top_p.data(), batch_size, stream_);
    }

    dim3 block(std::min((int) batch_size, 256));
    dim3 grid(divUp((int) batch_size, (int) block.x));
    // support top_k up to 1024.
    setup_topk_runtime_args<1024><<<grid, block, 0, stream_>>>(batch_size, top_k, runtime_top_k_buf_,
        runtime_top_k_size, top_p, runtime_top_p_buf_, runtime_top_p_size, skip_decode_buf_);
    cudaAutoCpy(skip_decode_, skip_decode_buf_, batch_size, stream_);
    std::vector<uint32_t> runtime_top_ks(batch_size);
    cudaAutoCpy(runtime_top_ks.data(), runtime_top_k_buf_, batch_size, stream_);
    runtime_max_top_k_ = *std::max_element(std::begin(runtime_top_ks), std::end(runtime_top_ks));
}

template <typename T>
void TopKSamplingLayer<T>::runSampling(DecodingOutputParams& outputs, DecodingParams const& params)
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);

    auto const batch_size = outputs.output_ids.shape[1];
    auto const local_batch_size = params.logits.shape[0];
    auto const ite = params.ite;
    auto const step = params.step;

    // in case of skip any, the logit value is already copied and processed.
    auto* logits = !skip_any_ ? params.logits.template getPtr<T>() : runtime_logits_buf_;
    auto* end_ids = params.end_ids.template getPtr<const int>();

    bool* finished = (outputs.finished) ? outputs.finished->template getPtr<bool>() : nullptr;
    invokeAddBiasEndMask(
        logits, (T*) (nullptr), end_ids, finished, local_batch_size, vocab_size_, vocab_size_padded_, stream_);
    sync_check_cuda_error();

    float* cum_log_probs = (outputs.cum_log_probs) ? outputs.cum_log_probs->template getPtr<float>() : nullptr;
    float* output_log_probs = (outputs.output_log_probs) ? outputs.output_log_probs->template getPtr<float>() : nullptr;

    if (cum_log_probs != nullptr || output_log_probs != nullptr)
    {
        invokeAddBiasSoftMax(
            logits, (T*) (nullptr), end_ids, finished, local_batch_size, vocab_size_padded_, vocab_size_, stream_);
        sync_check_cuda_error();
    }

    int* sequence_length = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;

    invokeBatchTopKSampling(sampling_workspace_, sampling_workspace_size_, logits,
        outputs.output_ids.getPtrWithOffset<int>(step * batch_size + ite * local_batch_size), sequence_length, finished,
        cum_log_probs, output_log_probs, curandstate_buf_ + ite * local_batch_size,
        (int) runtime_max_top_k_, // useless because runtime_top_k_buf_ is never
                                  // nullptr. Keep for legacy.
        (int*) (runtime_top_k_buf_ + ite * local_batch_size),
        1.0f,                     // useless because runtime_top_p_buf_ is never nullptr. Keep for
                                  // legacy.
        runtime_top_p_buf_ + ite * local_batch_size, vocab_size_padded_, end_ids, stream_, local_batch_size,
        skip_decode_buf_ + ite * local_batch_size);
    sync_check_cuda_error();
}

template <typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(size_t vocab_size, size_t vocab_size_padded, hipStream_t stream,
    cublasMMWrapper* cublas_wrapper, IAllocator* allocator, bool is_free_buffer_after_forward)
    : BaseSamplingLayer<T>(
        vocab_size, vocab_size_padded, stream, cublas_wrapper, allocator, is_free_buffer_after_forward, nullptr)
{
}

template <typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(TopKSamplingLayer<T> const& top_k_sampling_layer)
    : BaseSamplingLayer<T>(top_k_sampling_layer)
{
}

template <typename T>
TopKSamplingLayer<T>::~TopKSamplingLayer()
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
    freeBuffer();
}

template class TopKSamplingLayer<float>;
template class TopKSamplingLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
