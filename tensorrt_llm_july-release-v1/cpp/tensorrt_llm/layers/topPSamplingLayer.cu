#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/topPSamplingLayer.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

static __global__ void set_topp_runtime_args(int batch_size, std::uint32_t top_k, std::uint32_t* top_ks,
    int top_ks_size, float top_p, float* top_ps, int top_ps_size, bool* skip_decode, float* initial_top_p_buf,
    float* top_p_decay_buf, float* top_p_min_buf)
{
    /**
     * @brief Setup the runtime arguments for topp, broadcasting top_p to top_ps
              and top_k to top_ks, verifying value ranges of top_p_decay/top_p_min.
     *
     * \param batch_size
     * \param top_k
     * \param top_ks                [batch_size]
     * \param top_ks_size
     * \param top_p
     * \param top_ps                [batch_size]
     * \param top_ps_size
     * \param skip_decode           [batch_size]
     * \param initial_top_p_buf     [batch_size]
     * \param top_p_decay_buf       [batch_size]
     * \param top_p_min_buf         [batch_size]
     *
     */

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < batch_size; i += gridDim.x * blockDim.x)
    {
        std::uint32_t k = top_ks_size > 1 ? top_ks[i] : top_k;
        float p = top_ps_size > 1 ? top_ps[i] : top_p;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        top_ks[i] = k;
        // Clip p value if it is out of range. range = [0.0, 1.0].
        top_ps[i] = p < 0.0f ? 0.0f : (p > 1.0f ? 1.0f : p);
        if (p < 0.0f || p > 1.0f)
        {
            printf(
                "[WARNING] topp (%f) is out of range ([0.0, 1.0f]) for token %d"
                " clip to closest number %f.\n",
                p, i, top_ps[i]);
        }
        skip_decode[i] = k > 0;

        initial_top_p_buf[i] = top_ps[i];
        if (top_p_decay_buf[i] > 1.0f || top_p_decay_buf[i] <= 0.0f)
        {
            printf(
                "[WARNING] top_p_decay_buf (%f) is out of range ([0.0, 1.0f]) for "
                "token %d,"
                " change to 1.0f.\n",
                top_p_decay_buf[i], i);
            top_p_decay_buf[i] = 1.0f;
        }
        if (top_p_min_buf[i] > 1.0f || top_p_min_buf[i] <= 0.0f)
        {
            printf(
                "[WARNING] top_p_min_buf (%f) is out of range ([0.0, 1.0f]) for "
                "token %d,"
                " change to 0.5f.\n",
                top_p_min_buf[i], i);
            top_p_min_buf[i] = 0.5f;
        }
    }
}

template <typename T>
void TopPSamplingLayer<T>::allocateBuffer(std::size_t batch_size, std::vector<float> const& top_p)
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
    float const max_top_p = (top_p.size() > 0) ? *std::max_element(std::begin(top_p), std::end(top_p)) : 0.0f;
    invokeTopPSampling<T>(nullptr, // workspace
        sampling_workspace_size_, cub_temp_storage_size_,
        nullptr,                   // output_ids
        nullptr,                   // sequence_length
        nullptr,                   // finished_buffer
        nullptr,                   // cum_log_probs
        nullptr,                   // output_log_probs
        nullptr,                   // log_probs
        topp_id_vals_buf_, topp_offset_buf_, begin_topp_offset_buf_, curandstate_buf_, batch_size, vocab_size_padded_,
        nullptr, max_top_p, stream_, cuda_device_prop_, skip_decode_buf_);
    sampling_workspace_ = allocator_->reMalloc(sampling_workspace_, sampling_workspace_size_, true);
    runtime_top_k_buf_ = allocator_->reMalloc(runtime_top_k_buf_, sizeof(std::uint32_t) * batch_size, false);
    runtime_top_p_buf_ = allocator_->reMalloc(runtime_top_p_buf_, sizeof(float) * batch_size, false);
    initial_top_p_buf_ = allocator_->reMalloc(initial_top_p_buf_, sizeof(float) * batch_size, false);
    top_p_decay_buf_ = allocator_->reMalloc(top_p_decay_buf_, sizeof(float) * batch_size, false);
    top_p_min_buf_ = allocator_->reMalloc(top_p_min_buf_, sizeof(float) * batch_size, false);
    top_p_reset_ids_buf_ = allocator_->reMalloc(top_p_reset_ids_buf_, sizeof(std::int32_t) * batch_size, false);
    topp_id_vals_buf_
        = allocator_->reMalloc(topp_id_vals_buf_, sizeof(std::int32_t) * batch_size * vocab_size_padded_, false);
    topp_offset_buf_ = allocator_->reMalloc(topp_offset_buf_, sizeof(std::int32_t) * (batch_size + 1), false);
    begin_topp_offset_buf_
        = allocator_->reMalloc(begin_topp_offset_buf_, sizeof(std::int32_t) * (batch_size + 1), false);
    is_allocate_buffer_ = true;
}

template <typename T>
void TopPSamplingLayer<T>::freeBuffer()
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
    if (is_allocate_buffer_)
    {
        allocator_->free((void**) (&sampling_workspace_));
        allocator_->free((void**) (&topp_id_vals_buf_));
        allocator_->free((void**) (&topp_offset_buf_));
        allocator_->free((void**) (&begin_topp_offset_buf_));
        allocator_->free((void**) (&runtime_top_k_buf_));
        allocator_->free((void**) (&runtime_top_p_buf_));
        allocator_->free((void**) (&initial_top_p_buf_));
        allocator_->free((void**) (&top_p_decay_buf_));
        allocator_->free((void**) (&top_p_min_buf_));
        allocator_->free((void**) (&top_p_reset_ids_buf_));
    }
    BaseSamplingLayer<T>::freeBuffer();
    is_allocate_buffer_ = false;
}

template <typename T>
void TopPSamplingLayer<T>::setup(std::size_t const batch_size, SetupParams const& setupParams)
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
    BaseSamplingLayer<T>::setupBase(batch_size, setupParams);

    std::uint32_t const default_top_k = 0;
    auto const runtime_top_k = setupParams.runtime_top_k.value_or(std::vector<uint32_t>{default_top_k});
    auto const runtime_top_p = setupParams.runtime_top_p.value_or(std::vector<float>{});

    allocateBuffer(batch_size, runtime_top_p);

    std::size_t const runtime_top_k_size = runtime_top_k.size();
    std::size_t const runtime_top_p_size = runtime_top_p.size();

    if (runtime_top_p_size == 0)
    {
        std::fill_n(skip_decode_, batch_size, true);
        return;
    }

    std::uint32_t const top_k = runtime_top_k.at(0);
    float const top_p = runtime_top_p.at(0);

    if (runtime_top_k_size > 1)
    {
        TLLM_CHECK_WITH_INFO(runtime_top_k.size() == batch_size,
            fmtstr(
                "runtime_top_k.size() (%lu) == batch_size (%lu) is not satisfied!", runtime_top_k.size(), batch_size));
        cudaAutoCpy(runtime_top_k_buf_, runtime_top_k.data(), batch_size, stream_);
    }
    if (runtime_top_p_size > 1)
    {
        TLLM_CHECK_WITH_INFO(runtime_top_p.size() == batch_size,
            fmtstr(
                "runtime_top_p.size() (%lu) == batch_size (%lu) is not satisfied!", runtime_top_p.size(), batch_size));
        cudaAutoCpy(runtime_top_p_buf_, runtime_top_p.data(), batch_size, stream_);
    }

    auto fillBuffers = [this, &batch_size](std::string name, auto const& vector, auto& deviceBuffer)
    {
        TLLM_CHECK_WITH_INFO(vector.size() == batch_size,
            fmtstr("%s.size() (%lu) == batch_size (%lu) is not satisfied!", name.c_str(), vector.size(), batch_size));
        cudaAutoCpy(deviceBuffer, vector.data(), batch_size, stream_);
    };

    float const defaultTopPDecay{1.0f};
    fillBuffers("top_p_decay", setupParams.top_p_decay.value_or(std::vector<float>(batch_size, defaultTopPDecay)),
        top_p_decay_buf_);

    float const defaultTopPMin{1e-6f}; // prevent topp becoming 0.0
    fillBuffers(
        "top_p_min", setupParams.top_p_min.value_or(std::vector<float>(batch_size, defaultTopPMin)), top_p_min_buf_);

    std::int32_t const defaultTopPResetId{-1};
    fillBuffers("top_p_reset_ids",
        setupParams.top_p_reset_ids.value_or(std::vector<std::int32_t>(batch_size, defaultTopPResetId)),
        top_p_reset_ids_buf_);

    dim3 block(std::min((int) batch_size, 256));
    dim3 grid(divUp((int) batch_size, (int) block.x));
    set_topp_runtime_args<<<grid, block, 0, stream_>>>(batch_size, top_k, runtime_top_k_buf_, runtime_top_k_size, top_p,
        runtime_top_p_buf_, runtime_top_p_size, skip_decode_buf_, initial_top_p_buf_, top_p_decay_buf_, top_p_min_buf_);
    sync_check_cuda_error();

    cudaAutoCpy(skip_decode_, skip_decode_buf_, batch_size, stream_);

    std::vector<float> runtime_top_ps(batch_size);
    cudaAutoCpy(runtime_top_ps.data(), runtime_top_p_buf_, batch_size, stream_);
    runtime_max_top_p_ = *std::max_element(std::begin(runtime_top_ps), std::end(runtime_top_ps));
}

template <typename T>
void TopPSamplingLayer<T>::runSampling(DecodingOutputParams& outputs, DecodingParams const& params)
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);

    auto const batch_size = outputs.output_ids.shape[1];
    auto const local_batch_size = params.logits.shape[0];
    auto const ite = params.ite;
    auto const step = params.step;

    // in case of skip any, the logit value is already copied and processed.
    auto* logits = !skip_any_ ? params.logits.template getPtr<T>() : runtime_logits_buf_;
    auto* end_ids = params.end_ids.template getPtr<const int>();

    invokeTopPInitialize(
        topp_id_vals_buf_, topp_offset_buf_, begin_topp_offset_buf_, local_batch_size, vocab_size_padded_, stream_);
    sync_check_cuda_error();

    bool* finished = (outputs.finished) ? outputs.finished->template getPtr<bool>() : nullptr;
    invokeAddBiasSoftMax(
        logits, (T*) (nullptr), end_ids, finished, local_batch_size, vocab_size_padded_, vocab_size_, stream_);
    sync_check_cuda_error();

    float* cum_log_probs = (outputs.cum_log_probs) ? outputs.cum_log_probs->template getPtr<float>() : nullptr;
    float* output_log_probs = (outputs.output_log_probs) ? outputs.output_log_probs->template getPtr<float>() : nullptr;
    int* sequence_length = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;

    invokeBatchTopPSampling<T>(sampling_workspace_, sampling_workspace_size_, cub_temp_storage_size_,
        outputs.output_ids.getPtrWithOffset<int>(step * batch_size + ite * local_batch_size), sequence_length, finished,
        cum_log_probs, output_log_probs, logits, topp_id_vals_buf_, topp_offset_buf_, begin_topp_offset_buf_,
        curandstate_buf_ + ite * local_batch_size, local_batch_size, vocab_size_padded_, end_ids, runtime_max_top_p_,
        runtime_top_p_buf_ + ite * local_batch_size, stream_, cuda_device_prop_,
        skip_decode_buf_ + ite * local_batch_size);
    sync_check_cuda_error();

    invokeComputeToppDecay(runtime_top_p_buf_ + ite * local_batch_size, initial_top_p_buf_ + ite * local_batch_size,
        outputs.output_ids.getPtrWithOffset<int>(step * batch_size + ite * local_batch_size),
        top_p_decay_buf_ + ite * local_batch_size, top_p_min_buf_ + ite * local_batch_size,
        top_p_reset_ids_buf_ + ite * local_batch_size, local_batch_size, stream_);
    sync_check_cuda_error();
}

template <typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(std::size_t vocab_size, std::size_t vocab_size_padded, hipStream_t stream,
    cublasMMWrapper* cublas_wrapper, IAllocator* allocator, bool is_free_buffer_after_forward,
    hipDeviceProp_t* cuda_device_prop)
    : BaseSamplingLayer<T>(vocab_size, vocab_size_padded, stream, cublas_wrapper, allocator,
        is_free_buffer_after_forward, cuda_device_prop)
{
}

template <typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(TopPSamplingLayer<T> const& top_p_sampling_layer)
    : BaseSamplingLayer<T>(top_p_sampling_layer)
{
}

template <typename T>
TopPSamplingLayer<T>::~TopPSamplingLayer()
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
    freeBuffer();
}

template class TopPSamplingLayer<float>;
template class TopPSamplingLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
